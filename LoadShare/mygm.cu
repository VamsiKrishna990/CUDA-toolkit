#include "hip/hip_runtime.h"

# include "mygm.h"

__global__ void
myreduce_data(double *gid, double *god, unsigned int sizen)
{
    double *mysd = MySM<double>();
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    mysd[tid] = (i < sizen) ? gid[i] : 0;
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            if(mysd[tid] < mysd[tid + s])
                mysd[tid] = mysd[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) god[blockIdx.x] = mysd[0];
}
extern "C" double mygetmax(int n, int mean, int seed)
{
    int size = 1<<n; 
    int maxThreads = 256; 
    double maxvalue;
    unsigned int bytes = size * sizeof(double);
    double *h_idata = (double *) malloc(bytes);
    srand48(seed);
    for (int i=0; i<size; i++)
    {
        h_idata[i] = -mean * log(drand48());
    }
    int numBlocks = size / maxThreads; 
    int numThreads = size;
    int smemSize = maxThreads * sizeof(double);
    double *h_odata = (double *) malloc(numBlocks*sizeof(double));
    double  *d_idata = NULL;
    double *d_odata = NULL;
    double *d_intermediate = NULL;
    checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
    checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_intermediate, numBlocks*sizeof(double)));
    checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    myreduce_data<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_odata, numThreads);
    int s = numBlocks;
    while (s > 1) {
    checkCudaErrors(hipMemcpy(d_intermediate, d_odata, s * sizeof(double),
                                hipMemcpyDeviceToDevice));
        myreduce_data<<<(s+maxThreads-1)/maxThreads,maxThreads,smemSize>>>(d_intermediate, d_odata, s);
        s = (s+maxThreads-1)/maxThreads;
    }
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(double), hipMemcpyDeviceToHost));
    maxvalue = h_odata[0];
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    return maxvalue;
}
